#include "hip/hip_runtime.h"
/******************************************************************************
123456789_123456789_123456789_123456789_123456789_123456789_123456789_123456789_

					Introductory Graphics Library using GLUT/OpenGL 
					Northwestern University CS110 'Intro to Programming'

11/10/2003: Created --Andrew William Proksel
11/17/2003: Formatted, put fcn bodies in .c files, fixed order dependence, 
			changed floats to doubles, fixed float calls to int GL fcns,
			commented function internals, added more drawing and animation 
			prototypes. --Jack Tumblin
12/05/2003: Created function bodies for the prototypes, tested --A.W. Proksel
12/12/2003: Cleanup comments, revise disclaimer, added internal comments to
			all new functions measured angles from X axis, fixed sin/cos swap, 
			cleanup for consistent formatting, changed all floats to doubles 
			(again!), made #define for all angular steps in the .h file,
			removed redundant variables & openGL calls, added double-buffering
			(#define FINAL_GRADE  A) --J. Tumblin
4/12/2004:  Small clean-up by Prasun Choudhury. To be used in CS110 projects for
			Spring 2004.
8/20/2005:	Corrected and added comments.-- V. Doufexi

 ===============================================================================
  DISCLAIMER:
 introGlutLib.c and  introGlutLib.h originated as a student project written by
 Andrew William Proksel at Northwestern University, Evanston IL, as part of his
 classwork in the "Introduction to Computer Programming" course for non-majors 
 (CS110) taught by Jack Tumblin in Fall 2003.  It applies the wonderfully
  platform-independent 'GLUT' library written Mark Kilgard, ported to Win32 
 (Windows 95,98,Me,NT,2000,XP) by Nate Robins, and we thank them both for
 freely providing their software to everyone.  To obtain it, please visit:
	http://www.xmission.com/~nate/glut.html

  You are welcome to use and modify this code in any way you wish, but:
	1) Please include this disclaimer in any code that includes our work, and 
	please mention us in the credits or bibliography of published work
		(we're academics--publicity is the best reward).
	2) You must hold us harmless for any use you make of this code. Said in  
	plain English, this means you cannot sue us or Northwestern University for 
	any mistakes we made in this software, no matter how you decide to use it 
	or change it, and no matter how badly you were hurt because of it.
	3) This code is provided 'as-is'. While we will gladly answer reasonable
	questions about it, you cannot expect us to maintain the code for you,
	nor can we act as a customer-service desk that helps you debug your code.
*******************************************************************************/

#include "introGlutLib.h"				//holds all function prototypes.

/*=============================================================================
Function Bodies for startup/shutdown
==============================================================================*/

//------------------------------------------------------------------------------
// InitGraphics()
//
//  Do all initialization and setup needed to use this 'introductory graphics'
//  library. 

void InitGraphics()
{    
  char* args[] = {"foobar", 0};
  int one=1;
  // glutInit wants the main's args, but we don't need them for our programs
  // so we'll just pass it some junk.
  	glutInit(&one,args);
	srand( (int) time(NULL) );
	glutInitDisplayMode(GLUT_DOUBLE| GLUT_RGBA);
									// single buffering, use full 32-bit color
	glutInitWindowPosition(NU_SCREEN_XPOS, NU_SCREEN_YPOS);
	glutInitWindowSize(NU_SCREENWIDTH,NU_SCREENHEIGHT);
	glutCreateWindow("Fractal Zoomer");
									// Open a window (Microsoft Windows)
	glClearColor(0.5,0.5,0.5,0.0);	// Set the background color
	glColor3d(0.0,0.0,0.0);			// Set the default 'drawing pen' color
	glPointSize(1.0);				// Set the line width.

	glMatrixMode(GL_PROJECTION);	// Select world-to-camera transform:
	glLoadIdentity();				// initialize it, then set
	gluOrtho2D(0.0, (GLdouble)NU_SCREENWIDTH, 0.0, (GLdouble)NU_SCREENHEIGHT);
	// left, right,bottom top;
									// an orthographic camera matrix; treats
									// glVertex() values as pixel measurements.
	glutDisplayFunc(myDisplay);		// Register the fcn. GLUT calls for drawing
	glutKeyboardFunc(myKeyboard);	// Register the fcn. GLUT calls for keyboard input
	glutMouseFunc(myMouse);		// Register the fcn. GLUT calls for mouse input
	glutIdleFunc(myIdle);			// Register the fcn. GLUT calls when idle.

}

//------------------------------------------------------------------------------
// CloseGraphics()
//
//  Start the GLUT system, which manages the on-screen windows and makes calls 
// to our 'myDisplay()' function when that window needs to be redrawn.

void CloseGraphics()
{
	glutMainLoop();	
}

//------------------------------------------------------------------------------
// SetWindow(left, right, bottom, top)
//
// Called when users drag the corners of the display window to change its size.
// Changes the OpenGL world-to-camera coordinate transformation matrix.

void setWindow(double left, double right, double bottom, double top)
{
	glMatrixMode(GL_PROJECTION);			// Select the matrix to change,
	glLoadIdentity();						// clear it,
	gluOrtho2D(left, right, bottom, top);	// multiply by new coordinates.
}

//------------------------------------------------------------------------------
// setViewport(left, right, bottom, top)
//
// Change the OpenGL camera-to-screen coordinate system.

void setViewport(int left, int right, int bottom, int top)
{
	glViewport(left, bottom, right - left, top - bottom);
}

/*==============================================================================
		Function Bodies for on-screen drawing and animation
==============================================================================*/

//------------------------------------------------------------------------------
// SetPenColor(red, green, blue)
//
// Set the drawing color. red, green, blue range from 0.0 to 1.0.
// 
//   Commonly used colors:
//		(0.0,0.0,0.0) = black		(1.0,1.0,1.0) = white
//		(1.0,0.0,0.0) = red			(0.0,1.0,1.0) = cyan
//		(0.0,1.0,0.0) = green		(1.0,0.0,1.0) = magenta
//		(0.0,0.0,1.0) = blue		(1.0,1.0,0.0) = yellow
//		(0.5,0.5,0.5) = grey		(1.0,0.5,0.0) = orange
//		(0.5,0.0,0.5) = purple		(0.5,0.25,0.0) = brown
//		(0.0,0.25,0.0) = forest green
//		(0.0,0.0,0.25) = midnight blue

 
void SetPenColor(double red, double green, double blue)
{
	glColor3d(red,green,blue);
}
				
//------------------------------------------------------------------------------
// DrawLine(x0, y0, x1, y1)
// 
// Draw a line from (x0,y0) and ending at (x1,y1). 
// (doesn't use relative addressing).
// x,y are measured in pixels.

void DrawLine(double x0, double y0, double x1, double y1)
{
	glBegin(GL_LINES);				// Draw lines between pairs of points.
	glVertex2d(x0,y0);				// first line: begins at this point,
	glVertex2d(x1,y1);				// ends at this point.
	glEnd();						// No more pairs of points to draw.
	glFlush();						// Finish any pending drawing commands.
}

//-----------------------------------------------------------------------------
// DrawBox (x0, y0, x1, y1)
// 
// Draw a box using the current pen color with lower left corner at (x0,y0)
// and upper right corner at x1, y1.
// x,y are measured in pixels.

void DrawBox(double x0, double y0, double x1, double y1)
{
	glBegin(GL_LINE_STRIP);			// Draw a connected line from
	glVertex2d(x0,y0);				// corner to
	glVertex2d(x1,y0);				// corner to
	glVertex2d(x1,y1);				// corner to
	glVertex2d(x0,y1);				// corner to
	glVertex2d(x0,y0);				// corner, 
	glEnd();						// then stop--we're finished.
	glFlush();						// Finish any pending drawing commands.
}

//------------------------------------------------------------------------------
// DrawFillBox (x0, y0, x1, y1)
// 
// Draw a box and fill the region inside it using the current pen color. 
// You can specify the pen color with the SetPenColor() function.
// The default color is black.
// x,y are measured in pixels.

void DrawFillBox(double x0, double y0, double x1, double y1)

{
	glBegin(GL_POLYGON);			// Draw a connected line from
	glVertex2d(x0,y0);				// corner to
	glVertex2d(x1,y0);				// lcorner to
	glVertex2d(x1,y1);				// corner to
	glVertex2d(x0,y1);				// corner to
	glVertex2d(x0,y0);				// corner, 
	glEnd();						// then stop--we're finished.
	glFlush();						// Finish any pending drawing commands.
}

//------------------------------------------------------------------------------
// DrawFillTriangle(x0, y0, x1, y1, x2, y2)
//
// Draw a triangle and fill the region inside it using the current pen color
// Default fill color is black, but you can change it with the SetPenColor() function.
// x,y are measured in pixels.

void DrawFillTriangle(double x0, double y0, double x1, double y1, double x2, double y2)

{
	glBegin(GL_POLYGON);			// Draw a connected line from
	glVertex2d(x0,y0);				// Vertex 0
	glVertex2d(x1,y1);				// Vertex 1
	glVertex2d(x2,y2);				// Vertex 2
	glVertex2d(x0,y0);				// corner, 
	glEnd();						// then stop--we're finished.
	glFlush();						// Finish any pending drawing commands.
}


//-----------------------------------------------------------------------------
// DrawEllipse(xctr, yctr, radius_x, radius_y)
//
// Draw the outline of an ellipse centered at (xctr,yctr), 
// with width  'radius_x'  and height 'radius_y' 
// using the current pen color and line width. Default pen color is black, but 
// the 'SetPenColor()' function can change it.
// Draws the circle as a sequence of very short,straight lines. 
// x,y, radius are measured in pixels.

void DrawEllipse(double xctr, double yctr, double radiusX, double radiusY) 
{
	double vectorX,vectorY;			// vector to a point on circle from its center
	double angle;					// Angle in radians from circle start point.

	glBegin(GL_LINE_STRIP);		// Tell OpenGL to draw a series of lines:
	for(angle=0; angle < 2.0*M_PI + NU_ANGLESTEP; angle+= NU_ANGLESTEP)			
	{								// (>2PI so that circle is always closed)
		vectorX= xctr + radiusX * cos(angle);	// set line endpoint
		vectorY= yctr + radiusY * sin(angle);		
		glVertex2d(vectorX,vectorY);	// plot the line endpoint.
	}
	glEnd();						// finished drawing line segments.
	glFlush();						// Finish any pending drawing commands
}

void DrawFillEllipse(double xctr, double yctr, double radiusX, double radiusY) 
{
	double vectorX0,vectorY0, vectorX1, vectorY1;			// vector to a point on circle from its center
	double angle;					// Angle in radians from circle start point.

	glBegin(GL_TRIANGLES);		// Tell OpenGL to draw a series of lines
	vectorX1 = xctr + radiusX;
	vectorY1 = yctr;
	for(angle=0; angle < 2.0*M_PI + NU_ANGLESTEP; angle+= NU_ANGLESTEP)			
	{								// (>2PI so that circle is always closed)
		vectorX0 = vectorX1;
		vectorY0 = vectorY1;
		vectorX1= xctr + radiusX * cos(angle);	// set line endpoint
		vectorY1= yctr + radiusY * sin(angle);		
		glVertex2d(xctr,yctr);		// plot the points of a triangle (CCW order)
		glVertex2d(vectorX0,vectorY0);	// center-->old pt-->new pt.
		glVertex2d(vectorX1,vectorY1);
	}
	glEnd();						// finished drawing line segments.
	glFlush();						// Finish any pending drawing commands
}

//-----------------------------------------------------------------------------
// DrawCircle(xctr, yctr, radius)
//
// Draw the outline of a circle centered at (xctr,yctr) and width of 'radius'. 
// using the current pen color and line width. Default pen color is black, but 
// the 'SetPenColor()' function can change it.
// Draws the circle as a sequence of very short,straight lines. 
// x,y, radius are measured in pixels.

void DrawCircle(double xctr, double yctr, double radius)
{
	double vectorX,vectorY;			// vector to a point on circle from its center
	double angle;					// Angle in radians from circle start point.

	glBegin(GL_LINE_STRIP);		// Tell OpenGL to draw a series of lines:
	for(angle=0; angle < 2.0*M_PI + NU_ANGLESTEP; angle+= NU_ANGLESTEP)			
	{								// (>2PI so that circle is always closed)
		vectorX= xctr + radius * cos(angle);	// set line endpoint
		vectorY= yctr + radius * sin(angle);		
		glVertex2d(vectorX,vectorY);	// plot the line endpoint.
	}
	glEnd();						// finished drawing line segments.
	glFlush();						// Finish any pending drawing commands

}

//----------------------------------------------------------------------------
// DrawFillCircle(xctr, yctr, radius)
//
// Draw a circle and fill the region inside it using the the current pen color.  
// Default pen color is black, but the SetFillColor() function will change it.
// Draws the shape using filled triangles.
// x,y, radius are measured in pixels

void DrawFillCircle(double xctr, double yctr, double radius)
{
	double vectorX1,vectorY1;		// vector to a point on circle from its center
	double vectorX0,vectorY0;		// previous version of vectorX1,Y1;
	double angle;					// Angle in radians from circle start point.

	glBegin(GL_TRIANGLES);		// Tell OpenGL to draw a series of triangles
	vectorX1 = xctr + radius;	// Start at the circle's rightmost point.
	vectorY1 = yctr;		
	for(angle=NU_ANGLESTEP;		// step through all other points on circle;
		angle < 2.0*M_PI + NU_ANGLESTEP; angle+= NU_ANGLESTEP)			
	{								// (>2PI so that circle is always closed)
		vectorX0 = vectorX1;		// save previous point's position,
		vectorY0 = vectorY1;
		vectorX1= xctr + radius*cos(angle);	// find a new point on the circle,
		vectorY1= yctr + radius*sin(angle);		
		glVertex2d(xctr,yctr);		// plot the points of a triangle (CCW order)
		glVertex2d(vectorX0,vectorY0);	// center-->old pt-->new pt.
		glVertex2d(vectorX1,vectorY1);
	}
	glEnd();						// finished drawing triangles.
	glFlush();						// Finish any pending drawing commands
}

//-----------------------------------------------------------------------------
// DrawArc (xctr, yctr, radius, startAngle, endAngle)
//
// Draw a circular arc using the current pen color. Plots a portion of the
// circle whose center is (xctr,yctr) and width is given by 'radius', but only
// draws the portion of the circle between 'startAngle' and 'endAngle'.  Both
// the start and end angles are measured from the right-most point on the circle
// (e.g. the x-axis intercept for a circle centered at the origin) in the 
// counter-clockwise direction, in degrees.  Default pen color is black, but 
// the 'SetPenColor()' function can change it for you.
// Draws the arc as a sequence of short, straight-line segments.
// x,y, radius are measured in pixels
// Angles are measured in degrees

void DrawArc (double xctr, double yctr, double radius, double startAngle, double endAngle)
{
	double vectorX,vectorY;			// vector to a point on circle from its center
	double angle, ang0,ang1;

	ang0 = startAngle * (M_PI/180.0);	// convert degrees to radians
	ang1 = endAngle * (M_PI/180.0);
	glBegin(GL_LINE_STRIP);		// tell OpenGL to draw connected lines.
	for(angle=ang0; angle <= ang1+NU_ANGLESTEP; angle+= NU_ANGLESTEP)
		{
			vectorX = xctr + radius*cos(angle);	// find a line endpoint
			vectorY = yctr + radius*sin(angle);
			glVertex2d(vectorX,vectorY);	// plot that line endpoint,
		}
	glEnd();						// Finished drawing connected lines.
	glFlush();						// Finish any pending drawing commands
} 

//-----------------------------------------------------------------------------
// DrawPieArc (xctr, yctr, radius, startAngle, endAngle)
//
// Draw a circular arc and fill the pie-slice-shaped region between the arc 
// and its center point with the current pen color.  Default fill color is black, 
// but the SetFillColor() function can change it.
// Draws the shape using filled triangles.
// x,y, radius are measured in pixels
// Angles are measured in degrees

void DrawPieArc(double xctr, double yctr, double radius, double startAngle, double endAngle)
{
	double vectorX1,vectorY1;		// vector to a point on circle from its center
	double vectorX0,vectorY0;		// previous version of vectorX1,Y1;
	double angle,ang0,ang1;			// Angle in radians from circle start point.

	ang0 = startAngle * (M_PI/180.0);	// convert degrees to radians
	ang1 = endAngle * (M_PI/180.0);
	glBegin(GL_TRIANGLES);		// Tell OpenGL to draw a series of triangles
								// Start at the circle's rightmost point.
	vectorX1 = xctr + radius*cos(ang0);	
	vectorY1 = yctr + radius*sin(ang0);
	for(angle=ang0 + NU_ANGLESTEP;// step through all other points on circle;
		angle < ang1 + NU_ANGLESTEP; angle += NU_ANGLESTEP)			
	{								// (add to ang1 to ensure arcs can close)
		vectorX0 = vectorX1;		// save previous point's position,
		vectorY0 = vectorY1;
		vectorX1= xctr + radius*cos(angle);	// find a new point on the circle,
		vectorY1= yctr + radius*sin(angle);		
		glVertex2d(xctr,yctr);		// plot the points of a triangle (CCW order)
		glVertex2d(vectorX0,vectorY0);	// center-->old pt-->new pt.
		glVertex2d(vectorX1,vectorY1);
	}
	glEnd();						// finished drawing triangles.
	glFlush();						// Finish any pending drawing commands


	vectorY1=yctr;					// Set starting point
	vectorX1=xctr;
}

//-----------------------------------------------------------------------------
// DrawText2D(font, x0, y0, pString)
//
//  Write the text string stored at 'pString' to the display screen using the
//  current pen color and the specified font.  
//  It places the lower left corner of the first line of text 
//  at position x0,y0.  
//	Example:  DrawText2D(helv18, 10,20,"Hello!");
//
// Available fonts:
// rom10 (TIMES_ROMAN size 10)
// rom24 (TIMES_ROMAN size 24)
// helv10 (HELVETICA size 10)
// helv12 (HELVETICA size 12)
// helv18 (HELVETICA size 18)

void DrawText2D(void * font, double x0, double y0, const char* pString) 		
{
	int i;//, imax;							// counter for characters.
	int		lines;							// counter for each 'newline' char.

	lines = 0;
	glRasterPos2d(x0, y0);			// set text's lower-left corner position
	//imax = 1000;						// limit the number of chars we print.
	for(i=0; pString[i] != '\0'; i++)		// for each char,
	{									
		if (pString[i] == '\n')						// is it a new-line?
		{
			lines++;								// count it, and
			glRasterPos2d(x0, y0-(lines * 18.0));	// move down to next line
		}
	glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, pString[i]);
	}
}

//----------------------------------------------------------------------------
// SetDottedLines()
//
//  Change pen to draw dotted lines instead of solid lines. All lines drawn 
//  after this call will be drawn dotted until you call the 'DrawSolidLines()' 
//  function.

void SetDottedLines(void)
{
	glLineStipple(4, 0xAAAA);		// Set OpenGL's fill pattern bits,
	glEnable(GL_LINE_STIPPLE);		// enable it,
	glFlush();						// and finish any pending drawing commands
}

//-----------------------------------------------------------------------------
// SetSolidLines()
//
//  Change the pen back to the default of drawing solid lines.  All lines drawn
//  after this call will be drawn solid until you call the 'DrawDottedLines()' 
//  function.

void SetSolidLines(void)
{
	glDisable(GL_LINE_STIPPLE);		// disable the fill pattern
	glFlush();
}

//-----------------------------------------------------------------------------
// SetLineWidth(wide)
//
//  Change the width (in pixels) of the lines drawn by the pen.  Default value is 3.0;
//  Careful! if you set 'wide' to zero, all lines drawn are invisible!

void SetLineWidth(double wide)
{
	glLineWidth((GLfloat)wide);		// (cast to OpenGL's float type)
}

//-----------------------------------------------------------------------------
// ClearWindow()
//
//  Calls OpenGL 'clear screen' function; fills the screen with background color.
//  You can change that by calling the SetBackgndColor() function

void ClearWindow(void)
{
	//glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glClear(GL_COLOR_BUFFER_BIT);
}


//-----------------------------------------------------------------------------
// SetBackgndColor(red, green, blue)
//
//  Sets the color used by 'ClearWindow()' function, and then fills the entire
// window with that color.

void SetBackgndColor(double red, double green, double blue)
{
		glClearColor(red, green, blue, 0.0);	// Set the background color
		glClear(GL_COLOR_BUFFER_BIT);			// Clear window using that color
}
	                                                        
//-----------------------------------------------------------------------------
// Pause(milliseconds)
//
//  Pause for 'milliseconds' seconds.  For best animation, pause should about
//  1/60th = 166 milliseconds or less between each frame. 	                                                        

void Pause(int milliseconds)
{                         // uses Win32's timing utilities:
#ifndef WIN32
  usleep((unsigned long) milliseconds);
#else  
  Sleep((DWORD)milliseconds);// cast to Win32's 'DWORD' (long int) type
#endif
}
 
//-----------------------------------------------------------------------------
// myIdle is a function that allows for animation. This function is called
// by the InitGraphics(); Thus, it should not be called any place else.

void myIdle (void)
{
	glutSwapBuffers();
	glutPostRedisplay();
} 
