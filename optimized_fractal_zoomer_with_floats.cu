#include "hip/hip_runtime.h"
/* This program demonstrates the use of the Glut library:
   -- It draws two objects (fish) of different sizes, at different locations on the drawing window.
   -- It accepts and reacts to mouse input,
   -- It accepts and reacts to keyboard input.
*/


#include "introGlutLib.h"		//include the basic drawing library
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>				// for malloc and free
#include <string.h>				// for strcpy and other string fcns.
#define PALETTES 14
#define FUNCTIONS 4
#define FILTERS 3
#define MANDELBROT 0
#define NEWTON4 1
#define SPIDER 2
#define LAMBDA 3


typedef struct {
    unsigned char red;
    unsigned char green;
    unsigned char blue;
} Color;

Color *palette;

int colors1[14][4] = {
  {12, 0, 10, 20},
  {12, 50, 100, 240},
  {12, 20, 3, 26},
  {12, 230, 60, 20},
  {12, 25, 10, 9},
  {12, 230, 170, 0},
  {12, 20, 40, 10},
  {12, 0, 100, 0},
  {12, 5, 10, 10},
  {12, 210, 70, 30},
  {12, 90, 0, 50},
  {12, 180, 90, 120},
  {12, 0, 20, 40},
  {12, 30, 70, 200}
};

int colors2[7][4] = { //Rainbow
  {14, 0, 0, 255},
  {14, 111, 0, 255},
  {14, 143, 0, 255},
  {14, 255, 0, 0},
  {14, 255, 127, 0},
  {14, 255, 255, 0},
  {14, 0, 255, 0}
};

int colors3[14][4] = { //Fire
  {10, 10, 0, 0},
  {10, 255, 20, 0},
  {10, 255, 51, 0},
  {10, 255, 111, 0},
  {10, 255, 141, 11},
  {10, 255, 166, 69},
  {10, 255, 185, 105},
  {10, 255, 201, 135},
  {10, 255, 213, 101},
  {10, 255, 223, 184},
  {10, 255, 231, 204},
  {10, 255, 238, 222},
  {10, 255, 244, 237},
  {10, 255, 255, 255},
};

int colors4[4][4] = {
  {10, 70, 0, 20}, 
  {10, 100, 0, 100}, 
  {14, 255, 0, 0}, 
  {10, 255, 200, 0} 
};

int colors5[4][4] = { //Green-White
  {8, 40, 70, 10}, 
  {9, 40, 170, 10}, 
  {6, 100, 255, 70}, 
  {8, 255, 255, 255}
};

int colors6[5][4] = { //Blue
  {12, 0, 0, 64},
  {12, 0, 0, 255}, 
  {10, 0, 255, 255}, 
  {12, 128, 255, 255}, 
  {14, 64, 128, 255}
};

int colors7[2][4] = {
  {16, 0, 0, 0}, 
  {32, 255, 255, 255}
};

int colors8[5][4] = { 
  {14, 12, 0, 0},
  {14, 77, 56, 56},
  {14, 69, 6, 6},
  {14, 148, 55, 56},
  {14, 251, 195, 199}
};


int colors9[4][4] = { 
  {12, 84, 54, 4},
  {12, 116, 35, 7},
  {12, 233, 216, 127},
  {12, 127, 171, 233},
};

int colors10[5][4] = { 
  {12, 59, 90, 58},
  {12, 121, 157, 116},
  {12, 178, 157, 72},
  {12, 211, 191, 111},
  {12, 66, 42, 15}
};

int colors11[5][4] = { 
  {12, 242, 249, 209},
  {12, 254, 213, 121},
  {12, 199, 55, 31},
  {12, 176, 0, 57},
  {12, 53, 12, 26}
};

int colors12[5][4] = { 
  {12, 228, 253, 127},
  {12, 242, 176, 0},
  {12, 171, 33, 33},
  {12, 102, 25, 25},
  {12, 252, 92, 13}
};

int colors13[5][4] = { 
  {12, 214, 198, 146},
  {12, 241, 225, 202},
  {12, 96, 105, 62},
  {12, 129, 170, 102},
  {12, 86, 69, 41}
};

int colors14[5][4] = {   
  {12, 161, 36, 32},
  {12, 32, 15, 8},
  {12, 214, 207, 191},
  {12, 209, 184, 127},
  {12, 164, 117, 49}
};

float edge_kernel[25] = {
  -1.0, -1.0, -2.0, -1.0, -1.0,
  -1.0, -2.0, -4.0, -2.0, -1.0,
  -2.0, -4.0, 44.0, -4.0, -2.0,
  -1.0, -2.0, -4.0, -2.0, -1.0,
  -1.0, -1.0, -2.0, -1.0, -1.0
};

/*float edge_kernel[9] = {
  -1.0, -1.0, -1.0,
  -1.0,  8.0, -1.0,
  -1.0, -1.0, -1.0
};*/

float antialiasing_kernel[25] = {
  0.5625/12/12, 0.5625/12/12, 0.5625/12/12, 0.5625/12/12, 0.5625/12/12,
  0.5625/12/12,    0.5625/12,    0.5625/12,    0.5625/12, 0.5625/12/12,
  0.5625/12/12,    0.5625/12,       0.5625,    0.5625/12, 0.5625/12/12,
  0.5625/12/12,    0.5625/12,    0.5625/12,    0.5625/12, 0.5625/12/12,
  0.5625/12/12, 0.5625/12/12, 0.5625/12/12, 0.5625/12/12, 0.5625/12/12
};

struct BMPHeader {
    char bfType[2];       /* "BM" */
    int bfSize;           /* Size of file in bytes */
    int bfReserved;       /* set to 0 */
    int bfOffBits;        /* Byte offset to actual bitmap data (= 54) */
    int biSize;           /* Size of BITMAPINFOHEADER, in bytes (= 40) */
    int biWidth;          /* Width of image, in pixels */
    int biHeight;         /* Height of images, in pixels */
    short biPlanes;       /* Number of planes in target device (set to 1) */
    short biBitCount;     /* Bits per pixel (24 in this case) */
    int biCompression;    /* Type of compression (0 if no compression) */
    int biSizeImage;      /* Image size, in bytes (0 if no compression) */
    int biXPelsPerMeter;  /* Resolution in pixels/meter of display device */
    int biYPelsPerMeter;  /* Resolution in pixels/meter of display device */
    int biClrUsed;        /* Number of colors in the color table (if 0, use 
                             maximum allowed by biBitCount) */
    int biClrImportant;   /* Number of important colors.  If 0, all colors 
                             are important */
};


typedef struct {
    unsigned char red;
    unsigned char green;
    unsigned char blue;
} Image;

Image image[IMAGE_SIZE * IMAGE_SIZE];

struct timeval calc_start, calc_end;
struct timeval filter_start, filter_end;



int max_iterations;
int tile_size;
int function;
int filter;
int active_palette;
int palette_size;
float xCenter;
float yCenter;
float size;
char calc_time[50];
char spercent_calc[50];
char filter_time[25];
char siterations[15];
char center[100];
char ssize[50];
char window_title[100];
int not_calculated;
int calculating = 0;
Image *d_image;
Image *d_image_out;
Color *d_palette;
float *d_kernel;
int *d_not_calculated;

void (*ptr_function)(Image*, Color*, int, int, float, float, float, int) = NULL;
void (*ptr_function2)(Image*, Color*, int, int, float, float, float, int, int, int*) = NULL;



// ========================== Function prototypes:============================
void myDisplay();
void myMouse(int button, int state, int x, int y);
void myKeyboard(unsigned char key, int x, int y);
void imageProcessing(float* kernel, int kernel_size);
void startingPosition(void);
void zoomIn(int x, int y);
void zoomOut(int x, int y);
int createPalette(int colors[][4], int colors_size);
void nextPalette(void);
void nextFunction(void);
void nextFilter(void);
void calculateFractal(float xCenter, float yCenter, float size, int max_iterations, int palette_size, int tile_size);
void saveBMPImage(void);
__global__ void MandelbrotGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size);
__global__ void Newton4GPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size);
__global__ void SpiderGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size);
__global__ void LambdaGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size);
__global__ void TiledMandelbrotGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size, int tile_size, int *d_not_calculated);
__global__ void TiledNewton4GPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size, int tile_size, int *d_not_calculated);
__global__ void TiledSpiderGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size, int tile_size, int *d_not_calculated);
__global__ void TiledLambdaGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size, int tile_size, int *d_not_calculated);
__global__ void convolve2D(Image *in, Image *out, int image_size, float* kernel, int kernel_size);
__device__ void inline Mandelbrot(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float re, float im, int i, int j, int image_size);
__device__ void inline Newton4(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float re, float im, int i, int j, int image_size);
__device__ void inline Spider(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float re, float im, int i, int j, int image_size);
__device__ void inline Lambda(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float re, float im, int i, int j, int image_size);

// =================================Main:=======================================				
int main(int argc, char *argv[]) {
  
    if(argc < 6) {
        printf("\nToo few arguments.\n");
	printf("\n./project_part1 'function' 'iterations' palette' 'calculation algorithm' 'filter'\n");
	printf("\nfunction = 0 (Mandelbrot) or 1 (Newton 4) or 2 (Spider) or 3 (Lambda).\n");
	printf("\niterations > 0.\n");
	printf("\npalette = 0 (Palette 0) or 1 (Rainbow) or 2 (Fire) or 3 (Palette 3) or 4 (Green-White) or 5 (Blue) or 6 (Palette 6) or 7 (Palette 7) or 8 (Palette 8) or 9 (Palette 9) or 10 (Palette 11) or 12 (Palette 12) or 13 (Palette 13).\n");
	printf("\ncalculation algorithm = 0 (Simple algorithm), calculation algorithm > 0 (Tile size, be advised small sizes might lead to error)\n");
	printf("\nfilter = 0 (No Filter) or 1 (Antialiasing) or 2 (Edge Detection).\n\n");
	return 0;
    }
    else if(argc > 6) {
        printf("\nToo many arguments.\n");
	printf("\n./project_part1 'function' 'iterations' 'palette'  'calculation algorithm' 'filter'\n");
	printf("\nfunction = 0 (Mandelbrot) or 1 (Newton 4) or 2 (Spider) or 3 (Lambda).\n");
	printf("\niterations > 0.\n");
	printf("\npalette = 0 (Palette 0) or 1 (Rainbow) or 2 (Fire) or 3 (Palette 3) or 4 (Green-White) or 5 (Blue) or 6 (Palette 6) or 7 (Palette 7) or 8 (Palette 8) or 9 (Palette 9) or 10 (Palette 11) or 12 (Palette 12) or 13 (Palette 13).\n");
	printf("\ncalculation algorithm = 0 (Simple algorithm), calculation algorithm > 0 (Tile size, be advised small sizes might lead to error)\n");
	printf("\nfilter = 0 (No Filter) or 1 (Antialiasing) or 2 (Edge Detection).\n\n");
	return 0;
    }
     
    function = atoi(argv[1]);
    if(function < 0 || function > 3) {
        printf("\nThe function must be a number between 0 and 3.\n\n");
	return 0;
    }
    
    max_iterations = atoi(argv[2]);
    if(max_iterations < 2) {
        printf("\nThe iterations number must be a number > 1.\n\n");
	return 0;
    }
    
    active_palette = atoi(argv[3]);
    if(active_palette < 0 || active_palette > 13) {
        printf("\nThe palette, can only get values between 0 and 13.\n\n");
	return 0;
    }
    
    tile_size = atoi(argv[4]);
    if(tile_size < 0) {
        printf("\nCalculation algorithm must a number >= 0.\n\n");
	return 0;
    }
    
    filter = atoi(argv[5]);
    if(filter < 0 || filter > 2) {
        printf("\nFilter, can only get values between 0 and 2.\n\n");
	return 0;
    }
    
    printf("\n          q , Esc : quit\n                f : next function\n                p : next palette\n                t : next filter\n                1 : starting position\n                s : save bmp image\n                + : zoom in (old center)\n                - : zoom out (old center)\n left mouse click : zoom in (chosen center)\nright mouse click : zoom out (chosen center)\n\n");

    
    InitGraphics();			// initialize GLUT/OpenGL
    
    if(function == 0) {
        sprintf(window_title, "Fractal Zoomer [Function: Mandelbrot, Iterations: %d, Image Size: %d]", max_iterations, IMAGE_SIZE);
        glutSetWindowTitle(window_title);
	ptr_function = &MandelbrotGPU;	
	ptr_function2 = &TiledMandelbrotGPU;
    }
    else if(function == 1) {
        sprintf(window_title, "Fractal Zoomer [Function: Newton 4, Iterations: %d, Image Size: %d]", max_iterations, IMAGE_SIZE);
        glutSetWindowTitle(window_title);
	ptr_function = &Newton4GPU;
	ptr_function2 = &TiledNewton4GPU;
    }
    else if(function == 2) {
        sprintf(window_title, "Fractal Zoomer [Function: Spider, Iterations: %d, Image Size: %d]", max_iterations, IMAGE_SIZE);
        glutSetWindowTitle(window_title);
	ptr_function = &SpiderGPU;
	ptr_function2 = &TiledSpiderGPU;
    }
    else {
        sprintf(window_title, "Fractal Zoomer [Function: Lambda, Iterations: %d, Image Size: %d]", max_iterations, IMAGE_SIZE);
        glutSetWindowTitle(window_title); 
	ptr_function = &LambdaGPU;
	ptr_function2 = &TiledLambdaGPU;
    }
    
    switch(active_palette) {
        case 0:
	    palette_size = createPalette(colors1, 14);
	    break;
	case 1:
	    palette_size = createPalette(colors2, 7);
	    break;
	case 2:
	    palette_size = createPalette(colors3, 14);
	    break;
	case 3:
	    palette_size = createPalette(colors4, 4);
	    break;
	case 4:
	    palette_size = createPalette(colors5, 4);
	    break;
	case 5:
	    palette_size = createPalette(colors6, 5);
	    break;
	case 6:
	    palette_size = createPalette(colors7, 2);
	    break;
	case 7:
	    palette_size = createPalette(colors8, 5);
	    break;
	case 8:
	    palette_size = createPalette(colors9, 4);
	    break;
	case 9:
	    palette_size = createPalette(colors10, 5);
	    break;
	case 10:
	    palette_size = createPalette(colors11, 5);
	    break;
	case 11:
	    palette_size = createPalette(colors12, 5);
	    break;
	case 12:
	    palette_size = createPalette(colors13, 5);
	    break;
	case 13:
	    palette_size = createPalette(colors14, 5);
	    break;  
    }
	
    
    startingPosition();
    
    glutMainLoop();			// keep drawing
    
    free(palette);
    
    return 0;
    
}



/**********************************************************************
 myMouse(button, state, x, y)

 GLUT CALLBACK: Don't call this function in your program--GLUT does it.

 button: Which button was clicked. 
			Possible values are GLUT_LEFT for the left button and GLUT_RIGHT for the right one
 state: Is the button clicked (GLUT_DOWN) or not (GLUT_UP)?
 x, y: The coordinates of the place where you clicked (in pixels)
***********************************************************************/
			
void myMouse(int button, int state, int x, int y) {
	
	
	//int new_y = NU_SCREENHEIGHT-y;  // Even thought the normal output window has 
									// the origin point in the lower left corner, 
									// the mouse handling function assumes that
									// it's the upper left corner. 
									// So we change y to keep things consistent .

	
    if (state == GLUT_DOWN) {
		
		/* A button is being pressed. Set the correct motion function */
		
        if (button==GLUT_LEFT && !calculating) {
	    zoomIn(x, y);		
	}
	else if (!calculating) {
	    zoomOut(x, y);
	  
	}
		
    }
	
} 


/**********************************************************************
 myKeyboard(key, x, y)

 GLUT CALLBACK: Don't call this function in your program--GLUT does it.
***********************************************************************/

void myKeyboard(unsigned char key, int x, int y) {

    switch(key)  {
        case '=':
	    if(!calculating) {
	        zoomIn(IMAGE_SIZE / 2, IMAGE_SIZE / 2);
	    }
	    break;
	case '-':
	    if(!calculating) {
	        zoomOut(IMAGE_SIZE / 2, IMAGE_SIZE / 2);
	    }
	    break;
	case '1':
	    if(!calculating) {
	        startingPosition();
	    }
	    break;
	case 'p':
	    if(!calculating) {
	        nextPalette();
	    }
	    break;
	case 'f':
	    if(!calculating) {
	        nextFunction();
	    }
	    break;
	case 't':
	    if(!calculating) {
	        nextFilter();
	    }
	    break;
	case 's':
	    if(!calculating) {
	        saveBMPImage();
	    }
	    break;
        case 27:	// User pressed the Esc key 
	case 'Q':	// User pressed the Q key
	case 'q':   
	    exit(1);
	    break;
    }
	
}


/***************************************************************
 myDisplay()

 GLUT CALLBACK: Don't call this function in your program--GLUT does it.

 ######################################################################
 Students: put your drawing commands/function calls in this function, 
 rather than in main().
 ######################################################################
***************************************************************/

void myDisplay(void) {
  int x, y, temp_trans, temp_y, temp1, temp2, temp3, temp4, temp5;
    
    ClearWindow();
	
    SetBackgndColor(0.5, 0.5, 0.5);
 
    glBegin(GL_POINTS);
    for(y = 0; y < IMAGE_SIZE; y++) {
        temp_y = NU_SCREENHEIGHT - y;
        for(x = 0; x < IMAGE_SIZE; x++) {
            temp_trans = x * IMAGE_SIZE + y;
            SetPenColor(image[temp_trans].red / 255.0, image[temp_trans].green / 255.0, image[temp_trans].blue / 255.0);
            glVertex2d(x, temp_y);
        }
    }
    glEnd();
    glFlush();
	
    SetPenColor(0.0, 0.0, 0.0);
        
	
    temp1 = IMAGE_SIZE + 10;
    temp2 = IMAGE_SIZE - 130;
    temp3 = IMAGE_SIZE - 90;
    temp4 = IMAGE_SIZE - 50;
    temp5 = IMAGE_SIZE - 170;
    DrawText2D(helv10, temp1, temp4,  "Center:");
    if(-yCenter > 0) {
        sprintf(center, "%20.15lf+%20.15lfi", xCenter, -yCenter);	 
    }
    else if(yCenter == 0) {
        sprintf(center, "%20.15lf+%20.15lfi", xCenter, yCenter);
    }
    else {
        sprintf(center, "%20.15lf%20.15lfi", xCenter, -yCenter);
    }    
    DrawText2D(helv10, IMAGE_SIZE + 70, temp4, center);
	
    DrawText2D(helv10, temp1, temp3, "Size:");
    sprintf(ssize, "%20.15lf", size);
    DrawText2D(helv10, IMAGE_SIZE + 50, temp3, ssize);
		
    sprintf(spercent_calc, "(%3.2f%%)", (((float)IMAGE_SIZE * IMAGE_SIZE - not_calculated) / (IMAGE_SIZE * IMAGE_SIZE)) * 100);
    DrawText2D(helv10, temp1, temp2, spercent_calc);

    DrawText2D(helv10, IMAGE_SIZE + 100, temp2, "Calculation Time:");
    sprintf(calc_time, "%ld microseconds", (calc_end.tv_sec * 1000000 + calc_end.tv_usec) - (calc_start.tv_sec * 1000000 + calc_start.tv_usec));
    DrawText2D(helv10, IMAGE_SIZE + 260, temp2, calc_time);
	
    DrawText2D(helv10, temp1, temp5, "Filter Time:");    
    sprintf(filter_time, "%ld microseconds", (filter_end.tv_sec * 1000000 + filter_end.tv_usec) - (filter_start.tv_sec * 1000000 + filter_start.tv_usec));	
    DrawText2D(helv10, IMAGE_SIZE + 120, temp5, filter_time);
			
}


void startingPosition(void) {
  
    calculating = 1;
    
    if(function == LAMBDA) {
        xCenter = 1.0;
        yCenter = 0.0;
        size = 8.0;
    }
    else {
        xCenter = 0.0;
        yCenter = 0.0;
        size = 6.0;
    }
    
    
    calculateFractal(xCenter, yCenter, size, max_iterations, palette_size, tile_size);
    
    if(filter == 1) {
        imageProcessing(antialiasing_kernel, 5);
    }
    else if(filter == 2){
        imageProcessing(edge_kernel, 5);
    }
    else {
        gettimeofday(&filter_start, NULL);
        gettimeofday(&filter_end, NULL);    
    }
    
    printf("\n(%3.2f%%) Calculation time: %ld microseconds\n", ((((float)IMAGE_SIZE * IMAGE_SIZE - not_calculated) / (IMAGE_SIZE * IMAGE_SIZE)) * 100), (calc_end.tv_sec * 1000000 + calc_end.tv_usec) - (calc_start.tv_sec * 1000000 + calc_start.tv_usec));
    printf("Filter time: %ld microseconds\n", (filter_end.tv_sec * 1000000 + filter_end.tv_usec) - (filter_start.tv_sec * 1000000 + filter_start.tv_usec));
    
    calculating = 0;
    
}

void zoomIn(int x, int y) {
  
    calculating = 1;
    
    if(x < 0 || x >= IMAGE_SIZE || y < 0 || y >= IMAGE_SIZE) {
        calculating = 0;
        return;
    }
    

    xCenter = xCenter - size / 2 + size * x / IMAGE_SIZE;
    yCenter = yCenter - size / 2 + size * y / IMAGE_SIZE;
		    
    size /= 2;
		    
    calculateFractal(xCenter, yCenter, size, max_iterations, palette_size, tile_size);
    
    if(filter == 1) {
        imageProcessing(antialiasing_kernel, 5);
    }
    else if(filter == 2){
        imageProcessing(edge_kernel, 5);
    }
    else {
        gettimeofday(&filter_start, NULL);
        gettimeofday(&filter_end, NULL);    
    }
    
    printf("\n(%3.2f%%) Calculation time: %ld microseconds\n", ((((float)IMAGE_SIZE * IMAGE_SIZE - not_calculated) / (IMAGE_SIZE * IMAGE_SIZE)) * 100), (calc_end.tv_sec * 1000000 + calc_end.tv_usec) - (calc_start.tv_sec * 1000000 + calc_start.tv_usec));
    printf("Filter time: %ld microseconds\n", (filter_end.tv_sec * 1000000 + filter_end.tv_usec) - (filter_start.tv_sec * 1000000 + filter_start.tv_usec));
    
    calculating = 0;
    
}

void zoomOut(int x, int y) {
  
    calculating = 1;
    
    if(x < 0 || x >= IMAGE_SIZE || y < 0 || y >= IMAGE_SIZE) {
        calculating = 0;
        return;
    }
    
    
    xCenter = xCenter - size / 2 + size * x / IMAGE_SIZE;
    yCenter = yCenter - size / 2 + size * y / IMAGE_SIZE;
		    
    size *= 2;
		    
    calculateFractal(xCenter, yCenter, size, max_iterations, palette_size, tile_size);
    
    if(filter == 1) {
        imageProcessing(antialiasing_kernel, 5);
    }
    else if(filter == 2){
        imageProcessing(edge_kernel, 5);
    }
    else {
        gettimeofday(&filter_start, NULL);
        gettimeofday(&filter_end, NULL);    
    }
    
    printf("\n(%3.2f%%) Calculation time: %ld microseconds\n", ((((float)IMAGE_SIZE * IMAGE_SIZE - not_calculated) / (IMAGE_SIZE * IMAGE_SIZE)) * 100), (calc_end.tv_sec * 1000000 + calc_end.tv_usec) - (calc_start.tv_sec * 1000000 + calc_start.tv_usec));
    printf("Filter time: %ld microseconds\n", (filter_end.tv_sec * 1000000 + filter_end.tv_usec) - (filter_start.tv_sec * 1000000 + filter_start.tv_usec));
    
    calculating = 0;
    
}

void nextFunction(void) {
    
    function = (function + 1) % FUNCTIONS;
    
    if(function == 0) {
        sprintf(window_title, "Fractal Zoomer [Function: Mandelbrot, Iterations: %d, Image Size: %d]", max_iterations, IMAGE_SIZE);
        glutSetWindowTitle(window_title);
	ptr_function = &MandelbrotGPU;	
	ptr_function2 = &TiledMandelbrotGPU;
    }
    else if(function == 1) {
        sprintf(window_title, "Fractal Zoomer [Function: Newton 4, Iterations: %d, Image Size: %d]", max_iterations, IMAGE_SIZE);
        glutSetWindowTitle(window_title);
	ptr_function = &Newton4GPU;
	ptr_function2 = &TiledNewton4GPU;
    }
    else if(function == 2) {
        sprintf(window_title, "Fractal Zoomer [Function: Spider, Iterations: %d, Image Size: %d]", max_iterations, IMAGE_SIZE);
        glutSetWindowTitle(window_title);
	ptr_function = &SpiderGPU;
	ptr_function2 = &TiledSpiderGPU;
    }
    else {
        sprintf(window_title, "Fractal Zoomer [Function: Lambda, Iterations: %d, Image Size: %d]", max_iterations, IMAGE_SIZE);
        glutSetWindowTitle(window_title); 
	ptr_function = &LambdaGPU;
	ptr_function2 = &TiledLambdaGPU;
    }
    
    startingPosition();
    
}

void nextFilter(void) {
  
    calculating = 1;
    
    filter = (filter + 1) % FILTERS;
    
    calculateFractal(xCenter, yCenter, size, max_iterations, palette_size, tile_size);
    
    if(filter == 1) {
        imageProcessing(antialiasing_kernel, 5);
    }
    else if(filter == 2){
        imageProcessing(edge_kernel, 5);
    }
    else {
        gettimeofday(&filter_start, NULL);
        gettimeofday(&filter_end, NULL);    
    }
    
    printf("\n(%3.2f%%) Calculation time: %ld microseconds\n", ((((float)IMAGE_SIZE * IMAGE_SIZE - not_calculated) / (IMAGE_SIZE * IMAGE_SIZE)) * 100), (calc_end.tv_sec * 1000000 + calc_end.tv_usec) - (calc_start.tv_sec * 1000000 + calc_start.tv_usec));
    printf("Filter time: %ld microseconds\n", (filter_end.tv_sec * 1000000 + filter_end.tv_usec) - (filter_start.tv_sec * 1000000 + filter_start.tv_usec));
    
    calculating = 0;
}

void nextPalette(void) {
    
    calculating = 1;
    
    active_palette = (active_palette + 1) % PALETTES;
    
    free(palette);
    
    switch(active_palette) {
        case 0:
	    palette_size = createPalette(colors1, 14);
	    break;
	case 1:
	    palette_size = createPalette(colors2, 7);
	    break;
	case 2:
	    palette_size = createPalette(colors3, 14);
	    break;
	case 3:
	    palette_size = createPalette(colors4, 4);
	    break;
	case 4:
	    palette_size = createPalette(colors5, 4);
	    break;
	case 5:
	    palette_size = createPalette(colors6, 5);
	    break;
	case 6:
	    palette_size = createPalette(colors7, 2);
	    break;
	case 7:
	    palette_size = createPalette(colors8, 5);
	    break;
	case 8:
	    palette_size = createPalette(colors9, 4);
	    break;
	case 9:
	    palette_size = createPalette(colors10, 5);
	    break;
	case 10:
	    palette_size = createPalette(colors11, 5);
	    break;
	case 11:
	    palette_size = createPalette(colors12, 5);
	    break;
	case 12:
	    palette_size = createPalette(colors13, 5);
	    break;
	case 13:
	    palette_size = createPalette(colors14, 5);
	    break;	  
    }
    
    calculateFractal(xCenter, yCenter, size, max_iterations, palette_size, tile_size);
    
    if(filter == 1) {
        imageProcessing(antialiasing_kernel, 5);
    }
    else if(filter == 2){
        imageProcessing(edge_kernel, 5);
    }
    else {
        gettimeofday(&filter_start, NULL);
        gettimeofday(&filter_end, NULL);    
    }
    
    printf("\n(%3.2f%%) Calculation time: %ld microseconds\n", ((((float)IMAGE_SIZE * IMAGE_SIZE - not_calculated) / (IMAGE_SIZE * IMAGE_SIZE)) * 100), (calc_end.tv_sec * 1000000 + calc_end.tv_usec) - (calc_start.tv_sec * 1000000 + calc_start.tv_usec));
    printf("Filter time: %ld microseconds\n", (filter_end.tv_sec * 1000000 + filter_end.tv_usec) - (filter_start.tv_sec * 1000000 + filter_start.tv_usec));
    
    calculating = 0;
    
}


int createPalette(int colors[][4], int colors_size) {
  int i, n = 0, j;
  int *c1, *c2;
  
    for(i = 0; i < colors_size; i++) {
        n += colors[i][0];
    }
    
    palette = (Color *) malloc(sizeof(Color) * n);
    
    n = 0;
    for(i = 0; i < colors_size; i++) {
        c1 = colors[i];
	c2 = colors[(i + 1) % colors_size];
	for(j = 0; j < c1[0]; j++) {
	    palette[n + j].red = (c1[1] * (c1[0] - 1 - j) + c2[1] * j) / (c1[0] - 1);
	    palette[n + j].green = (c1[2] * (c1[0] - 1 - j) + c2[2] * j) / (c1[0] - 1);
	    palette[n + j].blue = (c1[3] * (c1[0] - 1 - j) + c2[3] * j) / (c1[0] - 1);
	}
	n += c1[0];
    }
    
    
    return n;
       
}

void calculateFractal(float xCenter, float yCenter, float size, int max_iterations, int palette_size, int tile_size) {
  int temp_break;
  hipError_t error;
  float temp_size_2;
  
    not_calculated = 0;
    temp_size_2 = size / 2;
  
    if(tile_size == 0) { 
        hipMalloc(&d_image, IMAGE_SIZE * IMAGE_SIZE * sizeof(Image));
        hipMalloc(&d_palette, palette_size * sizeof(Color));

        gettimeofday(&calc_start, NULL);
   
        hipMemcpy(d_palette, palette, palette_size * sizeof(Color), hipMemcpyHostToDevice);
    
        temp_break = IMAGE_SIZE >> 3;
        temp_break = temp_break == 0 ? 1 : temp_break;
        dim3 dimGrid(temp_break, temp_break);    
        dim3 dimBlock(IMAGE_SIZE / temp_break, IMAGE_SIZE / temp_break);
        ptr_function<<<dimGrid, dimBlock>>>(d_image, d_palette, max_iterations, palette_size, xCenter - temp_size_2 , yCenter - temp_size_2 , size / IMAGE_SIZE, IMAGE_SIZE);
    
        hipDeviceSynchronize();
    
        error = hipGetLastError();
    
        if(error != hipSuccess) {
            printf("\nCUDA Error: %s\n\n", hipGetErrorString(error));
	    hipFree(d_image);
            hipFree(d_palette);
	    return;
        }
    
        if(!filter) {
            hipMemcpy(&image, d_image, IMAGE_SIZE * IMAGE_SIZE * sizeof(Image), hipMemcpyDeviceToHost);
	    hipFree(d_image);
	}

        gettimeofday(&calc_end, NULL);
    
        hipFree(d_palette); 
    }
    else {
        hipMalloc(&d_image, IMAGE_SIZE * IMAGE_SIZE * sizeof(Image));
        hipMalloc(&d_palette, palette_size * sizeof(Color));
        hipMalloc(&d_not_calculated, sizeof(int));
      
        gettimeofday(&calc_start, NULL);
      
        hipMemcpy(d_palette, palette, palette_size * sizeof(Color), hipMemcpyHostToDevice);
        hipMemset(d_not_calculated, 0, sizeof(int));
    
        temp_break = tile_size >> 3;
        temp_break = temp_break == 0 ? 1 : temp_break;
        dim3 dimGrid(temp_break, temp_break);    
        dim3 dimBlock(tile_size / temp_break, tile_size / temp_break);
        ptr_function2<<<dimGrid, dimBlock>>>(d_image, d_palette, max_iterations, palette_size, xCenter - temp_size_2, yCenter - temp_size_2, size / IMAGE_SIZE, IMAGE_SIZE, tile_size, d_not_calculated);

        hipDeviceSynchronize();
    
        error = hipGetLastError();
    
        if(error != hipSuccess) {
            printf("\nCUDA Error: %s\n\n", hipGetErrorString(error));
	    hipFree(d_image);
            hipFree(d_palette);
	    hipFree(d_not_calculated);
	    return;
        }
    
        if(!filter) {
            hipMemcpy(&image, d_image, IMAGE_SIZE * IMAGE_SIZE * sizeof(Image), hipMemcpyDeviceToHost);
	    hipFree(d_image);
	}
        hipMemcpy(&not_calculated, d_not_calculated, sizeof(int), hipMemcpyDeviceToHost);

        gettimeofday(&calc_end, NULL);
    
        hipFree(d_palette);
        hipFree(d_not_calculated);
    }
     
}

void imageProcessing(float* kernel, int kernel_size) {
  int temp_break;
  hipError_t error;
  
    hipMalloc(&d_image_out, IMAGE_SIZE * IMAGE_SIZE * sizeof(Image));
    hipMalloc(&d_kernel, kernel_size * kernel_size * sizeof(float));
    
    gettimeofday(&filter_start, NULL);
    
    
    hipMemcpy(d_kernel, kernel, kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice);
       
    temp_break = IMAGE_SIZE >> 3;
    temp_break = temp_break == 0 ? 1 : temp_break;
    dim3 dimGrid(temp_break, temp_break);    
    dim3 dimBlock(IMAGE_SIZE / temp_break, IMAGE_SIZE / temp_break);
    convolve2D<<<dimGrid, dimBlock>>>(d_image, d_image_out, IMAGE_SIZE, d_kernel, kernel_size);
    
    hipDeviceSynchronize();
    
    error = hipGetLastError();
    
    if(error != hipSuccess) {
        printf("\nCUDA Error: %s\n\n", hipGetErrorString(error));
	hipFree(d_kernel);
        hipFree(d_image_out);
        return;
    }

    
    hipMemcpy(&image, d_image_out, IMAGE_SIZE * IMAGE_SIZE * sizeof(Image), hipMemcpyDeviceToHost);
    
 
    gettimeofday(&filter_end, NULL);
    

    hipFree(d_kernel);
    hipFree(d_image_out);
  
}

void saveBMPImage(void) {
  int i, j;
    int bytesPerLine;
    unsigned char *line;

    FILE *file;
    struct BMPHeader bmph;

    // The length of each line must be a multiple of 4 bytes 
    calculating = 1;

    bytesPerLine = (3 * (IMAGE_SIZE + 1) / 4) * 4;

    //strcpy(bmph.bfType, "BM");
    bmph.bfType[0] = 'B';
    bmph.bfType[1] = 'M';
    bmph.bfOffBits = 54;
    bmph.bfSize = bmph.bfOffBits + bytesPerLine * IMAGE_SIZE;
    bmph.bfReserved = 0;
    bmph.biSize = 40;
    bmph.biWidth = IMAGE_SIZE;
    bmph.biHeight = IMAGE_SIZE;
    bmph.biPlanes = 1;
    bmph.biBitCount = 24;
    bmph.biCompression = 0;
    bmph.biSizeImage = bytesPerLine * IMAGE_SIZE;
    bmph.biXPelsPerMeter = 0;
    bmph.biYPelsPerMeter = 0;
    bmph.biClrUsed = 0;       
    bmph.biClrImportant = 0; 

    file = fopen ("image.bmp", "wb");
    if(file == NULL) {  
       return;
    }
  
    fwrite(&bmph.bfType, 2, 1, file);
    fwrite(&bmph.bfSize, 4, 1, file);
    fwrite(&bmph.bfReserved, 4, 1, file);
    fwrite(&bmph.bfOffBits, 4, 1, file);
    fwrite(&bmph.biSize, 4, 1, file);
    fwrite(&bmph.biWidth, 4, 1, file);
    fwrite(&bmph.biHeight, 4, 1, file);
    fwrite(&bmph.biPlanes, 2, 1, file);
    fwrite(&bmph.biBitCount, 2, 1, file);
    fwrite(&bmph.biCompression, 4, 1, file);
    fwrite(&bmph.biSizeImage, 4, 1, file);
    fwrite(&bmph.biXPelsPerMeter, 4, 1, file);
    fwrite(&bmph.biYPelsPerMeter, 4, 1, file);
    fwrite(&bmph.biClrUsed, 4, 1, file);
    fwrite(&bmph.biClrImportant, 4, 1, file);
  
    line = (unsigned char *)malloc(bytesPerLine);
    if (line == NULL) {
        fprintf(stderr, "Can't allocate memory for BMP file.\n");
        return;
    }

    for (i = IMAGE_SIZE - 1; i >= 0; i--) {
        for (j = 0; j < IMAGE_SIZE; j++) {
	    line[3 * j + 2] = (char)((image[j * IMAGE_SIZE + i].red > 255 ? 255 : image[j * IMAGE_SIZE + i].red));
	    line[3 * j + 1] = (char)((image[j * IMAGE_SIZE + i].green > 255 ? 255 : image[j * IMAGE_SIZE + i].green)); 
            line[3 * j] = (char)((image[j * IMAGE_SIZE + i].blue > 255 ? 255 : image[j * IMAGE_SIZE + i].blue));      
        }
        fwrite(line, bytesPerLine, 1, file);
    }

    free(line);
    fclose(file);
    
    calculating = 0;

}

__global__ void convolve2D(Image* in, Image* out, int image_size, float* kernel, int kernel_size) {
  int i, j, k, l, p, t, temp_trans, temp_trans2, temp, temp2, temp3;
  float sum_red = 0, sum_green = 0, sum_blue = 0;
  
    i = threadIdx.x + blockDim.x * blockIdx.x;
    j = threadIdx.y + blockDim.y * blockIdx.y;
      
    temp = kernel_size >> 1;

    for(k = i - temp, p = 0; p < kernel_size; k++, p++) {
        if(k >= 0 && k < image_size) {
            temp2 = k * image_size;
            temp3 = p * kernel_size;
            for(l = j - temp, t = 0; t < kernel_size; l++, t++) {
	        if(l >= 0 && l < image_size) {
		    temp_trans = temp2 + l;
		    temp_trans2 = temp3 + t;
	            sum_red += in[temp_trans].red * kernel[temp_trans2];
		    sum_green += in[temp_trans].green * kernel[temp_trans2];
		    sum_blue += in[temp_trans].blue * kernel[temp_trans2];
	        }
	    }
	}
    }
  
    temp_trans = i * image_size + j;
    out[temp_trans].red = sum_red;
    out[temp_trans].green = sum_green;
    out[temp_trans].blue = sum_blue;
  
}

__global__ void MandelbrotGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size) {
  int i, j;
    
    i = threadIdx.x + blockDim.x * blockIdx.x;
    j = threadIdx.y + blockDim.y * blockIdx.y;
    
    Mandelbrot(d_image, d_palette, max_iterations, palette_size, xcenter_size_2 + temp_size_image_size * i, ycenter_size_2 + temp_size_image_size * j, i, j, image_size);
    
}

__global__ void Newton4GPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size) {
  int i, j;
    
    i = threadIdx.x + blockDim.x * blockIdx.x;
    j = threadIdx.y + blockDim.y * blockIdx.y;
    
    Newton4(d_image, d_palette, max_iterations, palette_size, xcenter_size_2 + temp_size_image_size * i, ycenter_size_2 + temp_size_image_size * j, i, j, image_size);
    
}

__global__ void SpiderGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size) {
  int i, j;
    
    i = threadIdx.x + blockDim.x * blockIdx.x;
    j = threadIdx.y + blockDim.y * blockIdx.y;
    
    Spider(d_image, d_palette, max_iterations, palette_size, xcenter_size_2 + temp_size_image_size * i, ycenter_size_2 + temp_size_image_size * j, i, j, image_size);
  
}

__global__ void LambdaGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size) {
  int i, j;
    
    i = threadIdx.x + blockDim.x * blockIdx.x;
    j = threadIdx.y + blockDim.y * blockIdx.y;
    
    Lambda(d_image, d_palette, max_iterations, palette_size, xcenter_size_2 + temp_size_image_size * i, ycenter_size_2 + temp_size_image_size * j, i, j, image_size);
    
}

__global__ void TiledMandelbrotGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size, int tile_size, int *d_not_calculated) {
  int x, y, i, j, tile_FROMx, tile_TOx, tile_FROMy, tile_TOy, whole_area, temp, temp1, temp2, temp3, temp4, temp5, k, l, step, temp_trans;
  float temp_y0, temp_x0;
  Color starting_color;
  __shared__ int s_not_calculated;
      
      s_not_calculated = 0;
  
      i = threadIdx.x + blockDim.x * blockIdx.x;
      j = threadIdx.y + blockDim.y * blockIdx.y;
  
      tile_FROMy = i * image_size / tile_size;
      tile_TOy = (i + 1) * image_size / tile_size;

      tile_FROMx = j * image_size / tile_size;
      tile_TOx = (j + 1) * image_size / tile_size;
	  
      temp = (tile_TOy - tile_FROMy + 1) / 2;
	     
      for(y = tile_FROMy, whole_area = 1, step = 0; step < temp; step++, whole_area = 1) {
          temp_y0 = ycenter_size_2 + temp_size_image_size * y;
	
	  x = tile_FROMx + step;
	  temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		
	  Mandelbrot(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		  
	  temp_trans = x * image_size + y;
	  starting_color.red = d_image[temp_trans].red;
	  starting_color.green = d_image[temp_trans].green;
	  starting_color.blue = d_image[temp_trans].blue;
		
	  for(; x < tile_TOx - step; x++) {  //FIRST ROW (moving right)
	      temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		    
	      Mandelbrot(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		      
	      temp_trans = x * image_size + y;
       	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }   
	  }
		
	  for(x--, y++; y < tile_TOy - step; y++) { //LAST COLUMN (moving down)
              temp_y0 = ycenter_size_2 + temp_size_image_size * y;
		     
       	      Mandelbrot(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }      
	  }
		
	  for(y--, x--; x >= tile_FROMx + step; x--) { //LAST ROW (moving left)
	      temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		    
	      Mandelbrot(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }     
	  }
	
	  for(x++, y--; y > tile_FROMy + step; y--) { //FIRST COLUMN (moving up)
	      temp_y0 = ycenter_size_2 + temp_size_image_size * y;
		    
	      Mandelbrot(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }    
	  }
		
	  y++;
		
	  if(whole_area) {
	      temp5 = step + 1;
	      temp1 = tile_TOx - temp5;
	      temp2 = tile_TOy - temp5;
	      temp3 = temp1 - x;
	      temp4 = temp2 - y;
		      
	      atomicAdd(&s_not_calculated, temp3 * temp4);
		    
	      for(k = y; k < temp2; k++) {
	          for(l = x + 1; l < temp1; l++) {
	    	      temp_trans = l * image_size + k;
		      d_image[temp_trans].red = starting_color.red;
		      d_image[temp_trans].green = starting_color.green;
		      d_image[temp_trans].blue = starting_color.blue;
		  }
	      }
		    
	      break;
	  }	
	
      }

      __syncthreads();

      if(threadIdx.x == 0 && threadIdx.y == 0) {
          atomicAdd(d_not_calculated, s_not_calculated);
      }
	
}


__global__ void TiledNewton4GPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size, int tile_size, int *d_not_calculated) {
  int x, y, i, j, tile_FROMx, tile_TOx, tile_FROMy, tile_TOy, whole_area, temp, temp1, temp2, temp3, temp4, temp5, k, l, step, temp_trans;
  float temp_y0, temp_x0;
  Color starting_color;
  __shared__ int s_not_calculated;
      
      s_not_calculated = 0;
  
      i = threadIdx.x + blockDim.x * blockIdx.x;
      j = threadIdx.y + blockDim.y * blockIdx.y;
  
      tile_FROMy = i * image_size / tile_size;
      tile_TOy = (i + 1) * image_size / tile_size;

      tile_FROMx = j * image_size / tile_size;
      tile_TOx = (j + 1) * image_size / tile_size;
	  
      temp = (tile_TOy - tile_FROMy + 1) / 2;
	     
      for(y = tile_FROMy, whole_area = 1, step = 0; step < temp; step++, whole_area = 1) {
          temp_y0 = ycenter_size_2 + temp_size_image_size * y;
	
	  x = tile_FROMx + step;
	  temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		
	  Newton4(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		  
	  temp_trans = x * image_size + y;
	  starting_color.red = d_image[temp_trans].red;
	  starting_color.green = d_image[temp_trans].green;
	  starting_color.blue = d_image[temp_trans].blue;
		
	  for(; x < tile_TOx - step; x++) {  //FIRST ROW (moving right)
	      temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		    
	      Newton4(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		      
	      temp_trans = x * image_size + y;
       	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }   
	  }
		
	  for(x--, y++; y < tile_TOy - step; y++) { //LAST COLUMN (moving down)
              temp_y0 = ycenter_size_2 + temp_size_image_size * y;
		     
       	      Newton4(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }      
	  }
		
	  for(y--, x--; x >= tile_FROMx + step; x--) { //LAST ROW (moving left)
	      temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		    
	      Newton4(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }     
	  }
	
	  for(x++, y--; y > tile_FROMy + step; y--) { //FIRST COLUMN (moving up)
	      temp_y0 = ycenter_size_2 + temp_size_image_size * y;
		    
	      Newton4(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }    
	  }
		
	  y++;
		
	  if(whole_area) {
	      temp5 = step + 1;
	      temp1 = tile_TOx - temp5;
	      temp2 = tile_TOy - temp5;
	      temp3 = temp1 - x;
	      temp4 = temp2 - y;
		      
	      atomicAdd(&s_not_calculated, temp3 * temp4);
		    
	      for(k = y; k < temp2; k++) {
	          for(l = x + 1; l < temp1; l++) {
	    	      temp_trans = l * image_size + k;
		      d_image[temp_trans].red = starting_color.red;
		      d_image[temp_trans].green = starting_color.green;
		      d_image[temp_trans].blue = starting_color.blue;
		  }
	      }
		    
	      break;
	  }	
	
      }

      __syncthreads();

      if(threadIdx.x == 0 && threadIdx.y == 0) {
          atomicAdd(d_not_calculated, s_not_calculated);
      }
	
}


__global__ void TiledSpiderGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size, int tile_size, int *d_not_calculated) {
  int x, y, i, j, tile_FROMx, tile_TOx, tile_FROMy, tile_TOy, whole_area, temp, temp1, temp2, temp3, temp4, temp5, k, l, step, temp_trans;
  float temp_y0, temp_x0;
  Color starting_color;
  __shared__ int s_not_calculated;
      
      s_not_calculated = 0;
  
      i = threadIdx.x + blockDim.x * blockIdx.x;
      j = threadIdx.y + blockDim.y * blockIdx.y;
  
      tile_FROMy = i * image_size / tile_size;
      tile_TOy = (i + 1) * image_size / tile_size;

      tile_FROMx = j * image_size / tile_size;
      tile_TOx = (j + 1) * image_size / tile_size;
	  
      temp = (tile_TOy - tile_FROMy + 1) / 2;
	     
      for(y = tile_FROMy, whole_area = 1, step = 0; step < temp; step++, whole_area = 1) {
          temp_y0 = ycenter_size_2 + temp_size_image_size * y;
	
	  x = tile_FROMx + step;
	  temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		
	  Spider(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		  
	  temp_trans = x * image_size + y;
	  starting_color.red = d_image[temp_trans].red;
	  starting_color.green = d_image[temp_trans].green;
	  starting_color.blue = d_image[temp_trans].blue;
		
	  for(; x < tile_TOx - step; x++) {  //FIRST ROW (moving right)
	      temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		    
	      Spider(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		      
	      temp_trans = x * image_size + y;
       	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }   
	  }
		
	  for(x--, y++; y < tile_TOy - step; y++) { //LAST COLUMN (moving down)
              temp_y0 = ycenter_size_2 + temp_size_image_size * y;
		     
       	      Spider(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }      
	  }
		
	  for(y--, x--; x >= tile_FROMx + step; x--) { //LAST ROW (moving left)
	      temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		    
	      Spider(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }     
	  }
	
	  for(x++, y--; y > tile_FROMy + step; y--) { //FIRST COLUMN (moving up)
	      temp_y0 = ycenter_size_2 + temp_size_image_size * y;
		    
	      Spider(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }    
	  }
		
	  y++;
		
	  if(whole_area) {
	      temp5 = step + 1;
	      temp1 = tile_TOx - temp5;
	      temp2 = tile_TOy - temp5;
	      temp3 = temp1 - x;
	      temp4 = temp2 - y;
		      
	      atomicAdd(&s_not_calculated, temp3 * temp4);
		    
	      for(k = y; k < temp2; k++) {
	          for(l = x + 1; l < temp1; l++) {
	    	      temp_trans = l * image_size + k;
		      d_image[temp_trans].red = starting_color.red;
		      d_image[temp_trans].green = starting_color.green;
		      d_image[temp_trans].blue = starting_color.blue;
		  }
	      }
		    
	      break;
	  }	
	
      }

      __syncthreads();

      if(threadIdx.x == 0 && threadIdx.y == 0) {
          atomicAdd(d_not_calculated, s_not_calculated);
      }
	
}


__global__ void TiledLambdaGPU(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float xcenter_size_2, float ycenter_size_2, float temp_size_image_size, int image_size, int tile_size, int *d_not_calculated) {
  int x, y, i, j, tile_FROMx, tile_TOx, tile_FROMy, tile_TOy, whole_area, temp, temp1, temp2, temp3, temp4, temp5, k, l, step, temp_trans;
  float temp_y0, temp_x0;
  Color starting_color;
  __shared__ int s_not_calculated;
      
      s_not_calculated = 0;
  
      i = threadIdx.x + blockDim.x * blockIdx.x;
      j = threadIdx.y + blockDim.y * blockIdx.y;
  
      tile_FROMy = i * image_size / tile_size;
      tile_TOy = (i + 1) * image_size / tile_size;

      tile_FROMx = j * image_size / tile_size;
      tile_TOx = (j + 1) * image_size / tile_size;
	  
      temp = (tile_TOy - tile_FROMy + 1) / 2;
	     
      for(y = tile_FROMy, whole_area = 1, step = 0; step < temp; step++, whole_area = 1) {
          temp_y0 = ycenter_size_2 + temp_size_image_size * y;
	
	  x = tile_FROMx + step;
	  temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		
	  Lambda(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		  
	  temp_trans = x * image_size + y;
	  starting_color.red = d_image[temp_trans].red;
	  starting_color.green = d_image[temp_trans].green;
	  starting_color.blue = d_image[temp_trans].blue;
		
	  for(; x < tile_TOx - step; x++) {  //FIRST ROW (moving right)
	      temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		    
	      Lambda(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		      
	      temp_trans = x * image_size + y;
       	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }   
	  }
		
	  for(x--, y++; y < tile_TOy - step; y++) { //LAST COLUMN (moving down)
              temp_y0 = ycenter_size_2 + temp_size_image_size * y;
		     
       	      Lambda(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }      
	  }
		
	  for(y--, x--; x >= tile_FROMx + step; x--) { //LAST ROW (moving left)
	      temp_x0 = xcenter_size_2 + temp_size_image_size * x;
		    
	      Lambda(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }     
	  }
	
	  for(x++, y--; y > tile_FROMy + step; y--) { //FIRST COLUMN (moving up)
	      temp_y0 = ycenter_size_2 + temp_size_image_size * y;
		    
	      Lambda(d_image, d_palette, max_iterations, palette_size, temp_x0, temp_y0, x, y, image_size);
		    
	      temp_trans = x * image_size + y;
	      if(d_image[temp_trans].red != starting_color.red || d_image[temp_trans].green != starting_color.green || d_image[temp_trans].blue != starting_color.blue) {
                  whole_area = 0;
    	      }    
	  }
		
	  y++;
		
	  if(whole_area) {
	      temp5 = step + 1;
	      temp1 = tile_TOx - temp5;
	      temp2 = tile_TOy - temp5;
	      temp3 = temp1 - x;
	      temp4 = temp2 - y;
		      
	      atomicAdd(&s_not_calculated, temp3 * temp4);
		    
	      for(k = y; k < temp2; k++) {
	          for(l = x + 1; l < temp1; l++) {
	    	      temp_trans = l * image_size + k;
		      d_image[temp_trans].red = starting_color.red;
		      d_image[temp_trans].green = starting_color.green;
		      d_image[temp_trans].blue = starting_color.blue;
		  }
	      }
		    
	      break;
	  }	
	
      }

      __syncthreads();

      if(threadIdx.x == 0 && threadIdx.y == 0) {
          atomicAdd(d_not_calculated, s_not_calculated);
      }
	
}

__device__ void inline Mandelbrot(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float re, float im, int i, int j, int image_size) {
  int iterations = 0, bailout = 4, temp_trans;
  float z_re = re, z_im = im, c_re = re, c_im = im, z_re_temp, squared_re, squared_im;
  int iterations_palette_size;
  unsigned char temp = 0;
  
    
    for(; iterations < max_iterations; iterations++) {
      
        squared_re = z_re * z_re;
	squared_im = z_im * z_im;
	
        if(squared_re + squared_im > bailout) {
	    break;
	}
	
	z_re_temp = squared_re - squared_im + c_re;
	z_im = 2 * z_re * z_im + c_im;
	z_re = z_re_temp;
	
    }
    
    temp_trans = i * image_size + j;
    iterations_palette_size = iterations % palette_size;
    temp -= (iterations != max_iterations);
    d_image[temp_trans].red = d_palette[iterations_palette_size].red & temp; 
    d_image[temp_trans].green = d_palette[iterations_palette_size].green & temp; 
    d_image[temp_trans].blue = d_palette[iterations_palette_size].blue & temp; 
    
}

__device__ void inline Newton4(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float re, float im, int i, int j, int image_size) {
  int iterations = 0, temp_trans;
  float epsilon = 1E-9;
  float z_re = re, z_im = im;
  float temp, temp2, temp3, divide_re, divide_im, dz_re, dz_im, old_re, old_im, temp_zre_oldre, temp_zim_old_im;
  int iterations_palette_size;
  unsigned char temp4 = 0;
  
  
    for(; iterations < max_iterations; iterations++) {
      
        if(iterations > 0 && (temp_zre_oldre = z_re - old_re) * (temp_zre_oldre) + (temp_zim_old_im = z_im - old_im) * (temp_zim_old_im) < epsilon) {
	    break;
	}
	
	old_re = z_re;
	old_im = z_im;
	
	temp = z_re * z_re;
	temp2 = z_im * z_im;

        dz_re = z_re * (temp - 3 * temp2);
        dz_im = z_im * (3 * temp - temp2);

        z_re = z_re * dz_re - z_im * dz_im - 1;
        z_im = old_re * dz_im + dz_re * z_im;

        dz_re *= 4;
        dz_im *= 4;
	
	temp3 = dz_re * dz_re + dz_im * dz_im;
	
	divide_re = (z_re * dz_re + z_im * dz_im) / temp3;
	divide_im = (z_im * dz_re - z_re * dz_im) / temp3;
	
	z_re = old_re - divide_re;
	z_im = old_im - divide_im;
    
    }
    
    temp_trans = i * image_size + j;
    iterations_palette_size = iterations % palette_size;
    temp4 -= (iterations != max_iterations);
    d_image[temp_trans].red = d_palette[iterations_palette_size].red & temp4; 
    d_image[temp_trans].green = d_palette[iterations_palette_size].green & temp4; 
    d_image[temp_trans].blue = d_palette[iterations_palette_size].blue & temp4; 
    
}

__device__ void inline Spider(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float re, float im, int i, int j, int image_size) {
  int iterations = 0, bailout = 4, temp_trans;
  float z_re = re, z_im = im, c_re = re, c_im = im, temp_re, squared_re, squared_im;
  int iterations_palette_size;
  unsigned char temp = 0;
 
  
    for(; iterations < max_iterations; iterations++) {
      
        squared_re = z_re * z_re;
	squared_im = z_im * z_im;
      
        if(squared_re + squared_im > bailout) {
	    break;
	}
	
	temp_re = squared_re - squared_im + c_re;
	z_im = 2 * z_re * z_im + c_im;
	z_re = temp_re;
	
	c_re = c_re / 2 + z_re;
	c_im = c_im / 2 + z_im;
	
    }
    
    temp_trans = i * image_size + j;
    iterations_palette_size = iterations % palette_size;
    temp -= (iterations != max_iterations);
    d_image[temp_trans].red = d_palette[iterations_palette_size].red & temp; 
    d_image[temp_trans].green = d_palette[iterations_palette_size].green & temp; 
    d_image[temp_trans].blue = d_palette[iterations_palette_size].blue & temp; 
  
}

__device__ void inline Lambda(Image *d_image, Color *d_palette, int max_iterations, int palette_size, float re, float im, int i, int j, int image_size) {
  int iterations = 0, bailout = 4, temp_trans;
  float z_re = 0.5, z_im = 0, c_re = re, c_im = im, temp_re, temp_im, temp_re2, temp_im2;
  int iterations_palette_size;
  unsigned char temp = 0;
  
  
    for(; iterations < max_iterations; iterations++) {
    
        if(z_re * z_re + z_im * z_im > bailout) {
	    break;
        }
      
        temp_re = c_re * z_re - c_im * z_im;
        temp_im = c_re * z_im + c_im * z_re;
      
        temp_re2 = 1 - z_re;
        temp_im2 = -z_im;
      
        z_re = temp_re * temp_re2 - temp_im * temp_im2;
        z_im = temp_re * temp_im2 + temp_im * temp_re2;
      
    } 
    
    temp_trans = i * image_size + j;
    iterations_palette_size = iterations % palette_size;
    temp -= (iterations != max_iterations);
    d_image[temp_trans].red = d_palette[iterations_palette_size].red & temp; 
    d_image[temp_trans].green = d_palette[iterations_palette_size].green & temp; 
    d_image[temp_trans].blue = d_palette[iterations_palette_size].blue & temp; 
    
}



